#include "hip/hip_runtime.h"
#include "faust_reduce_gpu.h"
#include <limits>

/*
#include "faust_cu_vec.h"
#include "faust_cu_matrix.h"

template<>
double faust_cu_reduce<double>(const faust_cu_vec<double>& v)
{
   thrust::device_ptr<double> dev_ptr(v.getData());
   double somme = thrust::reduce(dev_ptr, dev_ptr+v.size());
   return somme;
}

template<>
float faust_cu_reduce<float>(const faust_cu_vec<float>& v)
{
   thrust::device_ptr<float> dev_ptr(v.getData());
   float somme = thrust::reduce(dev_ptr, dev_ptr+v.size());
   return somme;
}


template<>
double faust_cu_reduce<double>(const faust_cu_matrix<double>& M)
{
   thrust::device_ptr<double> dev_ptr(M.getData());
   double somme = thrust::reduce(dev_ptr, dev_ptr+M.getNbRow()*M.getNbCol());
   return somme;
}

template<>
float faust_cu_reduce<float>(const faust_cu_matrix<float>& M)
{
   thrust::device_ptr<float> dev_ptr(M.getData());
   float somme = thrust::reduce(dev_ptr, dev_ptr+M.getNbRow()*M.getNbCol());
   return somme;
}*/


template<>
double faust_cu_sum<double>(const double* data, const int nb_el)
{
   thrust::device_ptr<const double> dev_ptr(data);
   const double somme = thrust::reduce(dev_ptr, dev_ptr+nb_el);
   return somme;
}
template<>
float faust_cu_sum<float>(const float* data, const int nb_el)
{
   thrust::device_ptr<const float> dev_ptr(data);
   const float somme = thrust::reduce(dev_ptr, dev_ptr+nb_el);
   return somme;
}

template<>
double faust_cu_max<double>(const double* data, const int nb_el)
{
   thrust::device_ptr<const double> dev_ptr(data);
   const double maxi = thrust::reduce(dev_ptr, dev_ptr+nb_el, -1.0e300,thrust::maximum<double>());
   return maxi;
}
template<>
float faust_cu_max<float>(const float* data, const int nb_el)
{
   thrust::device_ptr<const float> dev_ptr(data);
   const float maxi = thrust::reduce(dev_ptr, dev_ptr+nb_el, -1.0e300,thrust::maximum<float>());
   return maxi;
}

template<>
double faust_cu_min<double>(const double* data, const int nb_el)
{
   thrust::device_ptr<const double> dev_ptr(data);
   const double mini = thrust::reduce(dev_ptr, dev_ptr+nb_el, 1.0e300,thrust::minimum<double>());
   return mini;
}
template<>
float faust_cu_min<float>(const float* data, const int nb_el)
{
   thrust::device_ptr<const float> dev_ptr(data);
   const float mini = thrust::reduce(dev_ptr, dev_ptr+nb_el, 1.0e300,thrust::minimum<float>());
   return mini;
}

template<>
double faust_cu_norm<double>(const double* data, const int nb_el)
{
   thrust::device_ptr<const double> dev_ptr(data);
   const double frob_norm = std::sqrt(thrust::inner_product(dev_ptr, dev_ptr+nb_el, dev_ptr, 0.0, thrust::plus<double>() ,thrust::multiplies<double>()));
   return frob_norm;
}
template<>
float faust_cu_norm<float>(const float* data, const int nb_el)
{
   thrust::device_ptr<const float> dev_ptr(data);
   const float frob_norm = std::sqrt(thrust::inner_product(dev_ptr, dev_ptr+nb_el, dev_ptr, 0.0f, thrust::plus<float>() ,thrust::multiplies<float>()));
   return frob_norm;
}
